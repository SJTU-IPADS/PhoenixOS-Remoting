// https://gist.github.com/odashi/1c20ba90388cf02330e1b95963d78039

#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <chrono>

#include <hip/hip_runtime.h>
#include <cudnn.h>

#define likely(x) __builtin_expect(!!(x), 1)
#define unlikely(x) __builtin_expect(!!(x), 0)

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (unlikely(err != hipSuccess)) { \
    std::cout \
        << __FILE__ << ":" << __LINE__ << ": " << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

#define CUDNN_CALL(f) { \
  cudnnStatus_t err = (f); \
  if (unlikely(err != CUDNN_STATUS_SUCCESS)) { \
    std::cout \
        << __FILE__ << ":" << __LINE__ << ": " << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

__global__ void dev_const(float *px, float k) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = k;
}

__global__ void dev_iota(float *px) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = tid;
}

void print(const float *data, int n, int c, int h, int w) {
  std::vector<float> buffer(1 << 20);
  CUDA_CALL(hipMemcpy(
        buffer.data(), data,
        n * c * h * w * sizeof(float),
        hipMemcpyDeviceToHost));
  int a = 0;
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < c; ++j) {
      std::cout << "n=" << i << ", c=" << j << ":" << std::endl;
      for (int k = 0; k < h; ++k) {
        for (int l = 0; l < w; ++l) {
          std::cout << std::setw(4) << std::right << buffer[a];
          ++a;
        }
        std::cout << std::endl;
      }
    }
  }
  std::cout << std::endl;
}

int main() {
  cudnnHandle_t cudnn;
  CUDNN_CALL(cudnnCreate(&cudnn));

  // input
  const int in_n = 1;
  const int in_c = 1;
  const int in_h = 5;
  const int in_w = 5;
  std::cout << "in_n: " << in_n << std::endl;
  std::cout << "in_c: " << in_c << std::endl;
  std::cout << "in_h: " << in_h << std::endl;
  std::cout << "in_w: " << in_w << std::endl;
  std::cout << std::endl;

  cudnnTensorDescriptor_t in_desc;
  CUDNN_CALL(cudnnCreateTensorDescriptor(&in_desc));
  CUDNN_CALL(cudnnSetTensor4dDescriptor(
        in_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        in_n, in_c, in_h, in_w));

  float *in_data;
  CUDA_CALL(hipMalloc(
        &in_data, in_n * in_c * in_h * in_w * sizeof(float)));

  // filter
  const int filt_k = 1;
  const int filt_c = 1;
  const int filt_h = 2;
  const int filt_w = 2;
  std::cout << "filt_k: " << filt_k << std::endl;
  std::cout << "filt_c: " << filt_c << std::endl;
  std::cout << "filt_h: " << filt_h << std::endl;
  std::cout << "filt_w: " << filt_w << std::endl;
  std::cout << std::endl;

  cudnnFilterDescriptor_t filt_desc;
  CUDNN_CALL(cudnnCreateFilterDescriptor(&filt_desc));
  CUDNN_CALL(cudnnSetFilter4dDescriptor(
        filt_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW,
        filt_k, filt_c, filt_h, filt_w));

  float *filt_data;
  CUDA_CALL(hipMalloc(
      &filt_data, filt_k * filt_c * filt_h * filt_w * sizeof(float)));

  // convolution
  const int pad_h = 1;
  const int pad_w = 1;
  const int str_h = 1;
  const int str_w = 1;
  const int dil_h = 1;
  const int dil_w = 1;
  std::cout << "pad_h: " << pad_h << std::endl;
  std::cout << "pad_w: " << pad_w << std::endl;
  std::cout << "str_h: " << str_h << std::endl;
  std::cout << "str_w: " << str_w << std::endl;
  std::cout << "dil_h: " << dil_h << std::endl;
  std::cout << "dil_w: " << dil_w << std::endl;
  std::cout << std::endl;

  cudnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(cudnnCreateConvolutionDescriptor(&conv_desc));
  int padA[] = {pad_h, pad_w};
  int filterStrideA[] = {str_h, str_w};
  int dilationA[] = {dil_h, dil_w};
  CUDNN_CALL(cudnnSetConvolutionNdDescriptor(
        conv_desc, 2,
        padA, filterStrideA, dilationA,
        CUDNN_CONVOLUTION, CUDNN_DATA_FLOAT));

  int out[4];
  CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim(
        conv_desc, in_desc, filt_desc,
        4, out));
  // output
  int out_n;
  int out_c;
  int out_h;
  int out_w;

  out_n = out[0];
  out_c = out[1];
  out_h = out[2];
  out_w = out[3];

  std::cout << "out_n: " << out_n << std::endl;
  std::cout << "out_c: " << out_c << std::endl;
  std::cout << "out_h: " << out_h << std::endl;
  std::cout << "out_w: " << out_w << std::endl;
  std::cout << std::endl;

  cudnnTensorDescriptor_t out_desc;
  CUDNN_CALL(cudnnCreateTensorDescriptor(&out_desc));
  CUDNN_CALL(cudnnSetTensor4dDescriptor(
        out_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        out_n, out_c, out_h, out_w));

  float *out_data;
  CUDA_CALL(hipMalloc(
        &out_data, out_n * out_c * out_h * out_w * sizeof(float)));

  // algorithm
  cudnnConvolutionFwdAlgoPerf_t algo_perf;
  algo_perf.algo = CUDNN_CONVOLUTION_FWD_ALGO_GEMM;
//   int requestedAlgoCount = 1;
//   int returnedAlgoCount = 0;
//   CUDNN_CALL(cudnnFindConvolutionForwardAlgorithm(
//         cudnn,
//         in_desc, filt_desc, conv_desc, out_desc,
//         requestedAlgoCount, &returnedAlgoCount, &algo));

  std::cout << "Convolution algorithm: " << algo_perf.algo << std::endl;
  std::cout << std::endl;

  // workspace
  size_t ws_size;
  CUDNN_CALL(cudnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo_perf.algo, &ws_size));

  float *ws_data;
  CUDA_CALL(hipMalloc(&ws_data, ws_size));

  std::cout << "Workspace size: " << ws_size << std::endl;
  std::cout << std::endl;

  // perform
  float alpha = 1.f;
  float beta = 0.f;
  dev_iota<<<in_w * in_h, in_n * in_c>>>(in_data);
  dev_const<<<filt_w * filt_h, filt_k * filt_c>>>(filt_data, 1.f);

  // remove initial overhead
  for (int i = 0; i < 10; i++) {
    CUDNN_CALL(cudnnConvolutionForward(
        cudnn,
        &alpha, in_desc, in_data, filt_desc, filt_data,
        conv_desc, algo_perf.algo, ws_data, ws_size,
        &beta, out_desc, out_data));
  }

  const int numIterations = 100000;
  auto start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < numIterations; ++i) {
    CUDNN_CALL(cudnnConvolutionForward(
      cudnn,
      &alpha, in_desc, in_data, filt_desc, filt_data,
      conv_desc, algo_perf.algo, ws_data, ws_size,
      &beta, out_desc, out_data));
  }
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> elapsed = end - start;
  double totalElapsedTime = elapsed.count();
  // Calculate the average elapsed time
  double averageElapsedTime = totalElapsedTime / numIterations;
  std::cout << "Average elapsed time: " << averageElapsedTime << " ms" << std::endl;
  
  // results
  std::cout << "in_data:" << std::endl;
  print(in_data, in_n, in_c, in_h, in_w);
  
  std::cout << "filt_data:" << std::endl;
  print(filt_data, filt_k, filt_c, filt_h, filt_w);
  
  std::cout << "out_data:" << std::endl;
  print(out_data, out_n, out_c, out_h, out_w);

  // finalizing
  CUDA_CALL(hipFree(ws_data));
  CUDA_CALL(hipFree(out_data));
  CUDNN_CALL(cudnnDestroyTensorDescriptor(out_desc));
  CUDNN_CALL(cudnnDestroyConvolutionDescriptor(conv_desc));
  CUDA_CALL(hipFree(filt_data));
  CUDNN_CALL(cudnnDestroyFilterDescriptor(filt_desc));
  CUDA_CALL(hipFree(in_data));
  CUDNN_CALL(cudnnDestroyTensorDescriptor(in_desc));
  CUDNN_CALL(cudnnDestroy(cudnn));
  return 0;
}