#include "hip/hip_runtime.h"

#include <chrono>
#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char **argv)
{
    const int size = 1000000;
    const int iterations = 10000;
    int *a = new int[size];
    int *dev_a = nullptr;

    for (int i = 0; i < size; i++) {
        a[i] = i;
    }
    for (int i = 0; i < 10; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << std::endl;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void **)&dev_a, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

    // remove initial overhead
    for (int i = 0; i < 10; i++) {
        addKernel<<<2, (size + 1) / 2>>>(dev_a, dev_a, dev_a, size);
    }

    // Launch a kernel on the GPU with one thread for each element.
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < iterations; i++) {
        addKernel<<<2, (size + 1) / 2>>>(dev_a, dev_a, dev_a, size);
    }
    // cudaDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    // Calculate the elapsed time in milliseconds
    std::chrono::duration<double, std::milli> elapsed = end - start;

    double totalElapsedTime = elapsed.count();

    //  Calculate the average elapsed time
    double averageElapsedTime = totalElapsedTime / iterations;

    std::cout << "Total elapsed time: " << totalElapsedTime << " ms" << std::endl;
    std::cout << "Average elapsed time: " << averageElapsedTime << " ms" << std::endl;

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);

    for (int i = 0; i < 10; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
