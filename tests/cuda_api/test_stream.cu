#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
    // const int iterations = 1;
    // int count = 0;
    hipError_t error;

    error = hipStreamSynchronize(0);
    std::cout << "cudaStreamSynchronize(0) returned " << error << std::endl;

    hipStreamCaptureStatus status;
    error = hipStreamIsCapturing((ihipStream_t*)0, &status);
    std::cout << "cudaStreamIsCapturing(0) returned " << error << " and status " << status << std::endl;

    return 0;
}
