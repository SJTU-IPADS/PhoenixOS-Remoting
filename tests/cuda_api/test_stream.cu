#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
    const int iterations = 1;
    int count = 0;
    hipError_t error;

    error = hipStreamSynchronize(0);
    std::cout << "cudaStreamSynchronize(0) returned " << error << std::endl;

    hipStreamCaptureStatus status;
    hipStreamIsCapturing(0, &status);
    std::cout << "cudaStreamIsCapturing(0) returned " << status << std::endl;

    return 0;
}
