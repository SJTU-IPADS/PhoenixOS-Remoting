#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
    // const int iterations = 1;
    int count = 0;
    int device;

    hipGetDeviceCount(&count);
    std::cout << "Number of CUDA devices: " << count << std::endl;
    hipGetDevice(&device);
    std::cout << "Current CUDA device: " << device << std::endl;
    hipSetDevice(count - 1);
    hipGetDevice(&device);
    std::cout << "Current CUDA device: " << device << std::endl;

    // test cudaDeviceProp
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    std::cout << prop.name << std::endl;
    hipGetDeviceProperties(&prop, device);
    std::cout << prop.name << std::endl;

    // auto start = std::chrono::high_resolution_clock::now();
    // for (int i = 0; i < iterations; ++i) {
    //     cudaGetDevice(&device);
    //     cudaSetDevice(0);
    // }
    // auto end = std::chrono::high_resolution_clock::now();

    // // Calculate the elapsed time in milliseconds
    // std::chrono::duration<double, std::milli> elapsed = end - start;
    // double totalElapsedTime = elapsed.count();

    // //  Calculate the average elapsed time
    // double averageElapsedTime = totalElapsedTime / iterations;

    // std::cout << "Average elapsed time: " << averageElapsedTime << " ms" << std::endl;

    return 0;
}
