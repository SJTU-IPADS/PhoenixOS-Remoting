#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
    // const int iterations = 1;
    hipError_t error;

    error = hipGetLastError();
    std::cout << "Error: " << hipGetErrorString(error) << std::endl;

    error = hipPeekAtLastError();
    std::cout << "Error: " << hipGetErrorString(error) << std::endl;

    return 0;
}
