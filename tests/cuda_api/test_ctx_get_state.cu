#include <chrono>
#include <hip/hip_runtime.h>

#include <iostream>

int main()
{
    // get device
    int device = 0;
    hipGetDevice(&device);
    // call cuDevicePrimaryCtxGetState
    hipDevice_t dev = 0;
    unsigned int flags;
    int active;
    hipDevicePrimaryCtxGetState(dev, &flags, &active);
    std::cout << "flags: " << flags << ", active: " << active << std::endl;

    return 0;
}
