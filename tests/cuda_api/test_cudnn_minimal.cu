#include <hip/hip_runtime.h>
#include <cudnn.h>

#include <iostream>

/**
 * Minimal example to apply sigmoid activation on a tensor
 * using cuDNN.
 **/
int main(int argc, char** argv) {
  int numGPUs;
  hipGetDeviceCount(&numGPUs);
  std::cout << "Found " << numGPUs << " GPUs." << std::endl;
  hipSetDevice(0);  // use GPU0
  int device;
  struct hipDeviceProp_t devProp;
  hipGetDevice(&device);
  hipGetDeviceProperties(&devProp, device);
  std::cout << "Compute capability:" << devProp.major << "." << devProp.minor
            << std::endl;

  cudnnHandle_t handle_;
  cudnnCreate(&handle_); // ok
  std::cout << "Created cuDNN handle" << std::endl;

  // create the tensor descriptor
  cudnnDataType_t dtype = CUDNN_DATA_FLOAT;
  cudnnTensorFormat_t format = CUDNN_TENSOR_NCHW;
  int n = 1, c = 1, h = 1, w = 10;
  int NUM_ELEMENTS = n * c * h * w;
  cudnnTensorDescriptor_t x_desc;
  cudnnCreateTensorDescriptor(&x_desc); // ok
  cudnnSetTensor4dDescriptor(x_desc, format, dtype, n, c, h, w); // ok

  // create the tensor
  float* x;
  hipMallocManaged(&x, NUM_ELEMENTS * sizeof(float), hipMemAttachHost);
  for (int i = 0; i < NUM_ELEMENTS; i++) x[i] = i * 1.00f;
  std::cout << "Original array: ";
  for (int i = 0; i < NUM_ELEMENTS; i++) std::cout << x[i] << " ";

  // create activation function descriptor
  float alpha[1] = {1};
  float beta[1] = {0.0};
  cudnnActivationDescriptor_t sigmoid_activation;
  cudnnActivationMode_t mode = CUDNN_ACTIVATION_SIGMOID;
  cudnnNanPropagation_t prop = CUDNN_NOT_PROPAGATE_NAN;
  cudnnCreateActivationDescriptor(&sigmoid_activation); // ok
  cudnnSetActivationDescriptor(sigmoid_activation, mode, prop, 0.0f); // ok

  cudnnActivationForward(handle_, sigmoid_activation, alpha, x_desc, x, beta,
                         x_desc, x); // ?

  cudnnDestroy(handle_);
  std::cout << std::endl << "Destroyed cuDNN handle." << std::endl;
  std::cout << "New array: ";
  for (int i = 0; i < NUM_ELEMENTS; i++) std::cout << x[i] << " ";
  std::cout << std::endl;
  hipFree(x);
  return 0;
}